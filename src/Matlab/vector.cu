#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "mex.h"
/* Kernel to square elements of the array on the GPU */
__global__ void square_elements(float *in, float *out, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        out[idx] = in[idx] * in[idx];
}
/* Gateway function */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    int i, j, m, n;
    double *data1, *data2;
    float *data1f, *data2f;
    float *data1f_gpu, *data2f_gpu;
    mxClassID category;
    if (nrhs != nlhs)
        mexErrMsgTxt("The number of input and output arguments must be the same.");
    for (i = 0; i < nrhs; i++)
    {
        /* Find the dimensions of the data */
        m = mxGetM(prhs[i]);
        n = mxGetN(prhs[i]);
        /* Create an mxArray for the output data */
        plhs[i] = mxCreateDoubleMatrix(m, n, mxREAL);
        /* Create an input and output data array on the GPU*/
        hipMalloc((void **)&data1f_gpu, sizeof(float) * m * n);
        hipMalloc((void **)&data2f_gpu, sizeof(float) * m * n);
        /* Retrieve the input data */
        data1 = mxGetPr(prhs[i]);
        /* Check if the input array is single or double precision */
        category = mxGetClassID(prhs[i]);
        if (category == mxSINGLE_CLASS)
        {
            /* The input array is single precision, it can be sent directly to the card */
            hipMemcpy(data1f_gpu, data1, sizeof(float) * m * n, hipMemcpyHostToDevice);
        }
        if (category == mxDOUBLE_CLASS)
        {
            /* The input array is in double precision, it needs to be converted to floats before being sent to the card */
            data1f = (float *)mxMalloc(sizeof(float) * m * n);
            for (j = 0; j < m * n; j++)
            {
                data1f[j] = (float)data1[j];
            }
            hipMemcpy(data1f_gpu, data1f, sizeof(float) * n * m, hipMemcpyHostToDevice);
        }
        data2f = (float *)mxMalloc(sizeof(float) * m * n);
        /* Compute execution configuration using 128 threads per block */
        dim3 dimBlock(128);
        dim3 dimGrid((m * n) / dimBlock.x);
        if ((n * m) % 128 != 0)
            dimGrid.x += 1;
        /* Call function on GPU */
        square_elements<<<dimGrid, dimBlock>>>(data1f_gpu, data2f_gpu, n * m);
        /* Copy result back to host */
        hipMemcpy(data2f, data2f_gpu, sizeof(float) * n * m, hipMemcpyDeviceToHost);
        /* Create a pointer to the output data */
        data2 = mxGetPr(plhs[i]);
        /* Convert from single to double before returning */
        for (j = 0; j < m * n; j++)
        {
            data2[j] = (double)data2f[j];
        }
        /* Clean-up memory on device and host */
        mxFree(data1f);
        mxFree(data2f);
        hipFree(data1f_gpu);
        hipFree(data2f_gpu);
    }
}
