
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void kernel1(){
    printf("Hi Dragon\n");
}
extern "C" void cudaGet(int ompNumber,int mpiNumber){
    hipSetDevice(ompNumber+mpiNumber);
    kernel1<<<2,2>>>();
    hipDeviceSynchronize();
}