
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#define ElemType int
#define N 1'000'000
__global__ void kernel(const ElemType *a, const ElemType *b, ElemType *c)
{
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    c[id] = b[id] + a[id];
}
using namespace std;
int main(void)
{
    ElemType *device_variables[3];
    ElemType *host_variable = NULL;
    hipError_t status[3];
    int c = 2;
    host_variable = new int[N];
    if (host_variable == NULL)
    {
        fprintf(stderr, "malloc fail in host var\n");
        exit(1);
    }
    do
    {
        status[c--] = hipMalloc((void **)&device_variables[c], sizeof(ElemType) * N);
        if (status[c + 1] != hipSuccess)
        {
            fprintf(stderr, "malloc fail in device var\n");
            exit(2);
        }
    } while (c >= 0);
    for(int i=0;i<N;i++){
        host_variable[i] = 1;
    }
    hipMemcpy(device_variables[0], host_variable, sizeof(ElemType) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_variables[1], host_variable, sizeof(ElemType) * N, hipMemcpyHostToDevice);
    kernel<<<1000,1000>>>(device_variables[0],device_variables[1],device_variables[2]);
    hipMemcpy(host_variable, device_variables[2], sizeof(ElemType) * N, hipMemcpyDeviceToHost);
    for(int i=0;i<10;i++){
        printf("%d\n",host_variable[i]);
    }
    return 0;
}
